#include "hip/hip_runtime.h"
#include "path_planning/gpu_roi.hpp"
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <cstdio>
#include <cmath>
#include <limits>

// (선택) CUB 사용 시 더 빠름: #include <hipcub/hipcub.hpp>

namespace path_planning {

static inline void cudaCheck(hipError_t e, const char* where){
  if (e != hipSuccess) {
    std::fprintf(stderr, "[CUDA] %s: %s\n", where, hipGetErrorString(e));
  }
}

__global__ void transform_kernel(const float* __restrict__ x_in,
                                 const float* __restrict__ y_in,
                                 float* __restrict__ x_out,
                                 float* __restrict__ y_out,
                                 int N, float c, float s, float tx, float ty)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    float x = x_in[i], y = y_in[i];
    float xr = c * x - s * y + tx;
    float yr = s * x + c * y + ty;
    x_out[i] = xr; y_out[i] = yr;
  }
}

__global__ void distance_kernel(const float* __restrict__ x,
                                const float* __restrict__ y,
                                float* __restrict__ d, // d[0]=0
                                int N)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i == 0 && N > 0) d[0] = 0.0f;
  if (i > 0 && i < N) {
    float dx = x[i] - x[i-1];
    float dy = y[i] - y[i-1];
    d[i] = sqrtf(dx*dx + dy*dy);
  }
}

// 윈도우 내 x<=0은 큰 값(INF)로 취급하여 argmin에서 자동 제외
__global__ void squared_dist_window_xpos_kernel(const float* __restrict__ x,
                                                const float* __restrict__ y,
                                                float* __restrict__ d2,
                                                int N, int i0, int i1)
{
  int t = blockIdx.x * blockDim.x + threadIdx.x;
  int i = i0 + t;
  if (i < i1) {
    float xi = x[i];
    if (xi <= 0.0f) {
      d2[t] = INFINITY;
    } else {
      float yi = y[i];
      d2[t] = xi*xi + yi*yi;
    }
  }
}

__global__ void block_argmin_kernel(const float* __restrict__ vals,
                                    int M, // window length
                                    int* __restrict__ blk_idx,
                                    float* __restrict__ blk_val)
{
  extern __shared__ unsigned char smem[];
  float* s_val = reinterpret_cast<float*>(smem);
  int*   s_idx = reinterpret_cast<int*>(s_val + blockDim.x);

  int g = blockIdx.x * blockDim.x + threadIdx.x;

  float v = INFINITY; int idx = -1;
  if (g < M) { v = vals[g]; idx = g; }

  s_val[threadIdx.x] = v;
  s_idx[threadIdx.x] = idx;
  __syncthreads();

  for (int ofs = blockDim.x/2; ofs>0; ofs>>=1){
    if (threadIdx.x < ofs){
      if (s_val[threadIdx.x + ofs] < s_val[threadIdx.x]){
        s_val[threadIdx.x] = s_val[threadIdx.x + ofs];
        s_idx[threadIdx.x] = s_idx[threadIdx.x + ofs];
      }
    }
    __syncthreads();
  }
  if (threadIdx.x==0){
    blk_idx[blockIdx.x] = s_idx[0];
    blk_val[blockIdx.x] = s_val[0];
  }
}

struct GPURoiHelper::Impl {
  // 변환/아크 계산용 임시 버퍼(콜마다 할당/해제)
  // — transform_and_scan에서만 사용
  // 주기용(지속):
  thrust::device_vector<float> d_x;   // 변환된 x
  thrust::device_vector<float> d_y;   // 변환된 y

  // 윈도우 argmin용 버퍼(재사용)
  thrust::device_vector<float> d_win_d2;
  thrust::device_vector<int>   d_blk_idx;
  thrust::device_vector<float> d_blk_val;

  int N{0};

  // 비동기 결과 수신을 위한 스트림/이벤트/호스트버퍼
  hipStream_t stream{nullptr};
  hipEvent_t  evt_done{nullptr};
  int          last_i0{0};
  int          result_index{-1};
  bool         pending{false}; // 현재 진행 중인지

  Impl(){
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    hipEventCreateWithFlags(&evt_done, hipEventDisableTiming);
  }
  ~Impl(){
    if (evt_done) hipEventDestroy(evt_done);
    if (stream)   hipStreamDestroy(stream);
  }

  void clear_all(){
    d_x.clear(); d_y.clear();
    d_win_d2.clear(); d_blk_idx.clear(); d_blk_val.clear();
    N=0; pending=false; result_index=-1;
    last_i0=0;
  }
};

GPURoiHelper::GPURoiHelper() : pimpl_(new Impl) {}
GPURoiHelper::~GPURoiHelper(){ reset(); delete pimpl_; }

void GPURoiHelper::reset(){ pimpl_->clear_all(); }

bool GPURoiHelper::transform_and_scan(const std::vector<float>& x_in,
                                      const std::vector<float>& y_in,
                                      float c, float s, float tx, float ty,
                                      std::vector<float>& x_out,
                                      std::vector<float>& y_out,
                                      std::vector<double>& arc_out)
{
  const int N = static_cast<int>(x_in.size());
  if (N<=0 || y_in.size()!=static_cast<size_t>(N)) return false;

  thrust::device_vector<float> dx(N), dy(N), dxt(N), dyt(N), dseg(N);
  cudaCheck(hipMemcpy(thrust::raw_pointer_cast(dx.data()), x_in.data(), sizeof(float)*N, hipMemcpyHostToDevice), "Memcpy x_in");
  cudaCheck(hipMemcpy(thrust::raw_pointer_cast(dy.data()), y_in.data(), sizeof(float)*N, hipMemcpyHostToDevice), "Memcpy y_in");

  const int threads=256, blocks=(N+threads-1)/threads;
  transform_kernel<<<blocks,threads>>>(thrust::raw_pointer_cast(dx.data()),
                                       thrust::raw_pointer_cast(dy.data()),
                                       thrust::raw_pointer_cast(dxt.data()),
                                       thrust::raw_pointer_cast(dyt.data()),
                                       N, c, s, tx, ty);
  hipError_t e = hipDeviceSynchronize();
  if (e != hipSuccess) { std::fprintf(stderr,"[CUDA] transform failed: %s\n", hipGetErrorString(e)); return false; }

  distance_kernel<<<blocks,threads>>>(thrust::raw_pointer_cast(dxt.data()),
                                      thrust::raw_pointer_cast(dyt.data()),
                                      thrust::raw_pointer_cast(dseg.data()), N);
  e = hipDeviceSynchronize();
  if (e != hipSuccess) { std::fprintf(stderr,"[CUDA] distance failed: %s\n", hipGetErrorString(e)); return false; }

  thrust::inclusive_scan(dseg.begin(), dseg.end(), dseg.begin());

  x_out.resize(N); y_out.resize(N);
  std::vector<float> tmp_arc(N);
  cudaCheck(hipMemcpy(x_out.data(), thrust::raw_pointer_cast(dxt.data()), sizeof(float)*N, hipMemcpyDeviceToHost), "Memcpy x_out");
  cudaCheck(hipMemcpy(y_out.data(), thrust::raw_pointer_cast(dyt.data()), sizeof(float)*N, hipMemcpyDeviceToHost), "Memcpy y_out");
  cudaCheck(hipMemcpy(tmp_arc.data(), thrust::raw_pointer_cast(dseg.data()), sizeof(float)*N, hipMemcpyDeviceToHost), "Memcpy arc");

  arc_out.resize(N);
  for (int i=0;i<N;++i) arc_out[i] = static_cast<double>(tmp_arc[i]);
  return true;
}

void GPURoiHelper::upload_transformed_xy(const std::vector<float>& x_out,
                                         const std::vector<float>& y_out)
{
  const int N = static_cast<int>(x_out.size());
  pimpl_->d_x.assign(x_out.begin(), x_out.end());
  pimpl_->d_y.assign(y_out.begin(), y_out.end());
  pimpl_->N = N;
  pimpl_->d_win_d2.resize(N); // 최대 윈도우 길이를 N으로 할당(한 번만)
  pimpl_->pending = false;
  pimpl_->result_index = -1;
}

void GPURoiHelper::argmin_window_xpos_async(int last_idx, int W){
  if (pimpl_->N<=0) return;
  int N = pimpl_->N;
  int i0 = last_idx - W; if (i0<0) i0 = 0;
  int i1 = last_idx + W; if (i1>N) i1 = N;
  int M = i1 - i0;
  if (M<=0) { // 윈도우 비었으면 전체로
    i0 = 0; i1 = N; M = N;
  }
  pimpl_->last_i0 = i0;

  // 1) 거리계산(x>0 필터 포함)
  const int threads=256, blocks=(M+threads-1)/threads;
  squared_dist_window_xpos_kernel<<<blocks,threads,0,pimpl_->stream>>>(
      thrust::raw_pointer_cast(pimpl_->d_x.data()),
      thrust::raw_pointer_cast(pimpl_->d_y.data()),
      thrust::raw_pointer_cast(pimpl_->d_win_d2.data()),
      N, i0, i1);

  // 2) 블록 argmin
  int B = blocks;
  pimpl_->d_blk_idx.resize(B);
  pimpl_->d_blk_val.resize(B);
  size_t shmem = threads*(sizeof(float)+sizeof(int));

  block_argmin_kernel<<<B,threads,shmem,pimpl_->stream>>>(
      thrust::raw_pointer_cast(pimpl_->d_win_d2.data()), M,
      thrust::raw_pointer_cast(pimpl_->d_blk_idx.data()),
      thrust::raw_pointer_cast(pimpl_->d_blk_val.data()));

  // 3) 마지막 축소(작으니 호스트로 복사)
  //    결과 복사를 기다리기 위한 이벤트 등록
  hipEventRecord(pimpl_->evt_done, pimpl_->stream);
  pimpl_->pending = true;
}

bool GPURoiHelper::try_fetch_argmin_result(int& out_index){
  if (!pimpl_->pending) return false;
  hipError_t q = hipEventQuery(pimpl_->evt_done);
  if (q == hipSuccess){
    // 이벤트 완료: 블록 결과를 호스트로 가져와 최종 argmin
    int B = static_cast<int>(pimpl_->d_blk_idx.size());
    std::vector<int>   h_idx(B);
    std::vector<float> h_val(B);
    hipMemcpy(h_idx.data(), thrust::raw_pointer_cast(pimpl_->d_blk_idx.data()), sizeof(int)*B, hipMemcpyDeviceToHost);
    hipMemcpy(h_val.data(), thrust::raw_pointer_cast(pimpl_->d_blk_val.data()), sizeof(float)*B, hipMemcpyDeviceToHost);

    float best = INFINITY; int besti = -1;
    for (int b=0;b<B;++b){
      if (h_idx[b] >= 0 && h_val[b] < best){ best = h_val[b]; besti = h_idx[b]; }
    }
    if (besti >= 0){
      out_index = pimpl_->last_i0 + besti; // 윈도우 오프셋 보정
    } else {
      out_index = -1;
    }
    pimpl_->pending = false;
    pimpl_->result_index = out_index;
    return true;
  }
  // 아직 준비 안됨
  return false;
}

int GPURoiHelper::size() const { return pimpl_->N; }

} // namespace path_planning
